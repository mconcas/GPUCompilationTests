#include "hip/hip_runtime.h"
#include <cstdio>
#include "data.h"

extern __constant__ Data d_data;

extern "C" void runLibKernel();
__global__ void mainKernel()
{
    printf("Kernel on main executable -> a: %d, b: %f, c: %f\n", d_data.a, d_data.b, d_data.c);
}

void runMainKernel()
{
    mainKernel<<<1, 1>>>();
    hipDeviceSynchronize();
}

int main()
{
    Data h_data = {42, 3.14f, 2.718281828459045};
    hipMemcpyToSymbol(HIP_SYMBOL(d_data), &h_data, sizeof(Data));

    runLibKernel();
    runMainKernel();
    return 0;
}