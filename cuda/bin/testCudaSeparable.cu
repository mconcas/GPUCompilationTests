#include "hip/hip_runtime.h"
#include <cstdio>
#include "data.h"

template <typename T>
void discardResult(const T&)
{
}

extern __constant__ Data d_data;

extern "C" void runLibKernel();
__global__ void mainKernel()
{
    printf("Kernel on main executable -> a: %d, b: %f, c: %f\n", d_data.a, d_data.b, d_data.c);
}

void runMainKernel()
{
    mainKernel<<<1, 1>>>();
    discardResult(hipDeviceSynchronize());
}

int main()
{
    Data h_data = {42, 3.14f, 2.718281828459045};
    discardResult(hipMemcpyToSymbol(HIP_SYMBOL(d_data), &h_data, sizeof(Data)));

    runLibKernel();
    runMainKernel();
    return 0;
}