#include "hip/hip_runtime.h"
#include <cstdio>
#include "data.h"

template <typename T>
void discardResult(const T&)
{
}

__constant__ Data d_data;

__global__ void libKernel()
{
    printf("Kernel on library -> a: %d, b: %f, c: %f\n", d_data.a, d_data.b, d_data.c);
}

extern "C" void runLibKernel()
{
    libKernel<<<1, 1>>>();
    discardResult(hipDeviceSynchronize());
}