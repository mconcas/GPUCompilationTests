
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void testkernel() {
    printf("this is GPU!\n");
}

extern "C" void runLibKernel() {
    testkernel<<<1, 1>>>();
    hipDeviceSynchronize();
}